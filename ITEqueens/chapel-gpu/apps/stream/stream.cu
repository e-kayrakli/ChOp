
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <assert.h>
#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

__global__ void stream(float *dA, float *dB, float *dC, float alpha, int N) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < N) {
        dA[id] = dB[id] + alpha * dC[id];
    }
}

extern "C" {
    void streamCUDA(float* A, float *B, float *C, float alpha, int start, int end, int GPUN) {
        float *dA, *dB, *dC;
        if (GPUN > 0) {
            assert(end - start + 1 == GPUN);
#ifdef VERBOSE
            printf("In streamCUDA\n");
            printf("\t GPUN: %d\n", GPUN);
            printf("\t range: %d..%d\n", start, end);
#endif
            CudaSafeCall(hipMalloc(&dA, sizeof(float) * GPUN));
            CudaSafeCall(hipMalloc(&dB, sizeof(float) * GPUN));
            CudaSafeCall(hipMalloc(&dC, sizeof(float) * GPUN));

            CudaSafeCall(hipMemcpy(dB, B + start, sizeof(float) * GPUN, hipMemcpyHostToDevice));
            CudaSafeCall(hipMemcpy(dC, C + start, sizeof(float) * GPUN, hipMemcpyHostToDevice));

            stream<<<ceil(((float)GPUN)/1024), 1024>>>(dA, dB, dC, alpha, GPUN);

            CudaSafeCall(hipDeviceSynchronize());
            CudaSafeCall(hipMemcpy(A + start, dA, sizeof(float) * GPUN, hipMemcpyDeviceToHost));

            CudaSafeCall(hipFree(dA));
            CudaSafeCall(hipFree(dB));
            CudaSafeCall(hipFree(dC));
        }
    }
}
