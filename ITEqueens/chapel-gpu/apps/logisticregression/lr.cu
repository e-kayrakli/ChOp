
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <assert.h>
#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

__global__ void kernel1(float *dW, float *dWcurr, int N) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < N) {
        dWcurr[id] = dW[id];
    }
}

__global__ void kernel2(float *dW, float *dWcurr, float *dX, float *dY, float alpha, int nSamples, int nFeatures, int start, int N) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < N) {
        float err = 0.0;
        for (int s = 0; s < nSamples; s++) {
            float arg = 0.0;
            for (int f = 0; f < nFeatures; f++) {
                arg += dWcurr[f] * dX[s * (nFeatures) + f];
            }
            float hypo = 1 / (1 + exp(-arg));
            err += (hypo - dY[s]) * dX[s * (nFeatures) + start + id];
        }
        dW[id] = dWcurr[start + id] - alpha * err;
    }
}

extern "C" {
    void lrCUDA1(float *W, float *Wcurr, int start, int end, int GPUN) {
        float *dW, *dWcurr;
        if (GPUN > 0) {
            assert(end - start + 1 == GPUN);
#ifdef VERBOSE
            printf("In lrCUDA1\n");
            printf("\t GPUN: %d\n", GPUN);
            printf("\t range: %d..%d\n", start, end);
#endif
            CudaSafeCall(hipMalloc(&dW, sizeof(float) * GPUN));
            CudaSafeCall(hipMalloc(&dWcurr, sizeof(float) * GPUN));

            CudaSafeCall(hipMemcpy(dW, W + start, sizeof(float) * GPUN, hipMemcpyHostToDevice));
            kernel1<<<ceil(((float)GPUN)/1024), 1024>>>(dW, dWcurr, GPUN);

            CudaSafeCall(hipDeviceSynchronize());
            CudaSafeCall(hipMemcpy(Wcurr + start, dWcurr, sizeof(float) * GPUN, hipMemcpyDeviceToHost));

            CudaSafeCall(hipFree(dW));
            CudaSafeCall(hipFree(dWcurr));
        }
    }

    void lrCUDA2(float* X, float *Y, float *W, float *Wcurr, float alpha, int nSamples, int nFeatures, int start, int end, int GPUN) {
        float *dX, *dY, *dW, *dWcurr;
        if (GPUN > 0) {
            assert(end - start + 1 == GPUN);
#ifdef VERBOSE
            printf("In lrCUDA2\n");
            printf("\t GPUN: %d\n", GPUN);
            printf("\t range: %d..%d\n", start, end);
#endif
            CudaSafeCall(hipMalloc(&dX, sizeof(float) * nSamples * nFeatures));
            CudaSafeCall(hipMalloc(&dY, sizeof(float) * nSamples));
            CudaSafeCall(hipMalloc(&dWcurr, sizeof(float) * nFeatures));
            CudaSafeCall(hipMalloc(&dW, sizeof(float) * GPUN));

            CudaSafeCall(hipMemcpy(dX, X, sizeof(float) * nSamples * nFeatures, hipMemcpyHostToDevice));
            CudaSafeCall(hipMemcpy(dY, Y, sizeof(float) * nSamples, hipMemcpyHostToDevice));
            CudaSafeCall(hipMemcpy(dWcurr, Wcurr, sizeof(float) * nFeatures, hipMemcpyHostToDevice));

            kernel2<<<ceil(((float)GPUN)/1024), 1024>>>(dW, dWcurr, dX, dY, alpha, nSamples, nFeatures, start-1, GPUN);
            CudaSafeCall(hipDeviceSynchronize());
            CudaSafeCall(hipMemcpy(W, dW, sizeof(float) * GPUN, hipMemcpyDeviceToHost));

            CudaSafeCall(hipFree(dX));
            CudaSafeCall(hipFree(dY));
            CudaSafeCall(hipFree(dW));
            CudaSafeCall(hipFree(dWcurr));
        }
    }
}
