
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void myKernel(int64_t *dA, size_t nCols) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    int64_t *dA_row = (int64_t*)((char*)dA + i*nCols*sizeof(int64_t));
    dA_row[j] = dA_row[j] + 1;
}

extern "C" {
    void kernel(int64_t *ptr, size_t nRows, size_t nCols) {
        myKernel<<<nRows, nCols>>>(ptr, nCols);
    }
}