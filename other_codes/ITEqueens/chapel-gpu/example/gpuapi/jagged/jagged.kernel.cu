
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void myKernel(int64_t **dA) {
    for (int i = 0; i < 2; i++) {
        for (int j = 0; j < 256*(i+1); j++) {
            dA[i][j] = dA[i][j] + 1;
        }
    }
}

extern "C" {
    void kernelLOW(int64_t **hPtrs, size_t *hPtrSizes, int64_t N) {
        int64_t **dA = (int64_t**)malloc(sizeof(int64_t*)*N);
        for (int i = 0; i < N; i++) {
            hipMalloc(&dA[i], hPtrSizes[i]*sizeof(int64_t));
            hipMemcpy(dA[i], hPtrs[i], hPtrSizes[i]*sizeof(int64_t), hipMemcpyHostToDevice);
        }
        int64_t **dAs;
        hipMalloc(&dAs, sizeof(int64_t*)*N);
        hipMemcpy(dAs, dA, sizeof(int64_t*)*N, hipMemcpyHostToDevice);

        myKernel<<<1,1>>>(dAs);
        hipDeviceSynchronize();
        for (int i = 0; i < N; i++) {
            hipMemcpy(hPtrs[i], dA[i], hPtrSizes[i]*sizeof(int64_t), hipMemcpyDeviceToHost);
        }
    }

    void kernelMIDLOW(int64_t **dAs, int64_t N) {
        myKernel<<<1,1>>>(dAs);
        hipDeviceSynchronize();
    }
}