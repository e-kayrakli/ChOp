#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "../headers/GPU_queens.h"
#define _QUEENS_BLOCK_SIZE_ 128
#define _EMPTY_     -1
#define _MAX_ 32


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__device__  bool GPU_queens_stillLegal(const char *board, const int r){

  bool safe = true;
  int i;
  register int ld;
  register int rd;
  // Check vertical
  for ( i = 0; i < r; ++i)
    if (board[i] == board[r]) safe = false;
    // Check diagonals
    ld = board[r];  //left diagonal columns
    rd = board[r];  // right diagonal columns
    for ( i = r-1; i >= 0; --i) {
      --ld; ++rd;
      if (board[i] == ld || board[i] == rd) safe = false;
    }

    return safe;
}


__global__ void BP_queens_root_dfs(int N, unsigned int nPreFixos, int depthPreFixos,
    QueenRoot *root_prefixes,unsigned long long int *vector_of_tree_size, unsigned long long int *sols){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nPreFixos) {
        unsigned int flag = 0;
        unsigned int bit_test = 0;
        char board[32]; //representa o ciclo
        int N_l = N;
        int i, depth;
        unsigned long long  qtd_solucoes_thread = 0ULL;
        int depthGlobal = depthPreFixos;
        unsigned long long int tree_size = 0ULL;

        for (i = 0; i < N_l; ++i) {
            board[i] = _EMPTY_;
        }

        flag = root_prefixes[idx].control;


        for (i = 0; i < depthGlobal; ++i)
            board[i] = root_prefixes[idx].board[i];

        depth=depthGlobal;

        do{

            board[depth]++;
            bit_test = 0;
            bit_test |= (1<<board[depth]);

            if(board[depth] == N_l){
                board[depth] = _EMPTY_;
                //if(block_ub > upper)   block_ub = upper;
            }else if (!(flag &  bit_test ) && GPU_queens_stillLegal(board, depth)){

                    ++tree_size;
                    flag |= (1ULL<<board[depth]);

                    depth++;

                    if (depth == N_l) { //sol
                        ++qtd_solucoes_thread;
                    }else continue;
                }else continue;

            depth--;
            flag &= ~(1ULL<<board[depth]);

            }while(depth >= depthGlobal); //FIM DO DFS_BNB

        sols[idx] = qtd_solucoes_thread;
        vector_of_tree_size[idx] = tree_size;
    }//if
}//kernel
////////

extern "C" void GPU_call_cuda_queens(short size, int initial_depth, unsigned int n_explorers, QueenRoot *root_prefixes_h ,
	unsigned long long *vector_of_tree_size_h, unsigned long long *sols_h, int gpu_id){


//@todo -- use the Chapel GPU API By Akihiro
    hipSetDevice(gpu_id);
   // hipFuncSetCacheConfig(reinterpret_cast<const void*>(BP_queens_root_dfs),hipFuncCachePreferL1);


    unsigned long long *vector_of_tree_size_d;
    unsigned long long *sols_d;
    QueenRoot *root_prefixes_d;

    int num_blocks = ceil((double)n_explorers/_QUEENS_BLOCK_SIZE_);

    hipMalloc((void**) &vector_of_tree_size_d,n_explorers*sizeof(unsigned long long));
    hipMalloc((void**) &sols_d,n_explorers*sizeof(unsigned long long));
    hipMalloc((void**) &root_prefixes_d,n_explorers*sizeof(QueenRoot));


    //@todo -- use the Chapel GPU API By Akihiro
    printf("%d  - %d  - %d  -\n", num_blocks, n_explorers, _QUEENS_BLOCK_SIZE_ );
    hipMemcpy(root_prefixes_d, root_prefixes_h, n_explorers * sizeof(QueenRoot), hipMemcpyHostToDevice);

    BP_queens_root_dfs<<< num_blocks,_QUEENS_BLOCK_SIZE_>>> (size,n_explorers,initial_depth,root_prefixes_d, vector_of_tree_size_d,sols_d);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    hipMemcpy(vector_of_tree_size_h,vector_of_tree_size_d,n_explorers*sizeof(unsigned long long),hipMemcpyDeviceToHost);
    hipMemcpy(sols_h,sols_d,n_explorers*sizeof(unsigned long long),hipMemcpyDeviceToHost);

    hipFree(vector_of_tree_size_d);
    hipFree(sols_d);
    hipFree(root_prefixes_d);
    //After that, Chapel reduces the values
}
