
#include <hip/hip_runtime.h>
__global__ void vc(float *dA, float *dB, int N) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < N) {
        dA[id] = dB[id];
  }
}

extern "C" {
  void vcGPU(float* A, float *B, int start, int end, int GPUN) {
    float *dA, *dB;
    hipMalloc(&dA, sizeof(float) * GPUN);
    hipMalloc(&dB, sizeof(float) * GPUN);
    hipMemcpy(dB, B + start, sizeof(float) * GPUN, hipMemcpyHostToDevice);
    vc<<<ceil(((float)GPUN)/1024), 1024>>>(dA, dB, GPUN);
    hipDeviceSynchronize();
    hipMemcpy(A + start, dA, sizeof(float) * GPUN, hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
  }
}
